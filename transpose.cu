#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>


#define BLOCK 16


__global__
void parallelTransposeMemCoalescing(int* A, int* B, int m, int n) {
        __shared__ int block[BLOCK][BLOCK];

        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < m && j < n) {
                block[threadIdx.y][threadIdx.x] = A[i * n + j];
                __syncthreads();
                B[j * m + i] = block[threadIdx.y][threadIdx.x];
        }
}


__global__
void parallelTranspose(int* A, int* B, int m, int n) {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < m && j < n) {
                B[j * m + i] = A[i * n + j];
        }
}

int main(int argc, char *argv[]) {
        if (argc < 3) {
                printf("No enough arguments.");
                return -1;
        }

        srand(time(NULL));

        int m = atoi(argv[1]);
        int n = atoi(argv[2]);

        int* A = (int*) malloc(m * n * sizeof(int));
        int* B = (int*) malloc(m * n * sizeof(int));

        int i;
        for (i = 0; i < m * n; ++i)
                A[i] = rand() % 10;


        int *d_A, *d_B;
        hipMalloc(&d_A, n * m * sizeof(int));
        hipMalloc(&d_B, n * m * sizeof(int));

        //dimensions
        dim3 threadblock(BLOCK, BLOCK);
        dim3 grid(1 + n / threadblock.x, 1 + m / threadblock.y);

        //copying A to the GPU
        hipMemcpy(d_A, A, n * m * sizeof(int), hipMemcpyHostToDevice);





        /////////////////////////// FIRST EXECUTION ///////////////////////////
        clock_t t = clock();

        //calling function
        parallelTranspose<<<grid, threadblock>>>(d_A, d_B, m, n);
        hipDeviceSynchronize();

        //once the function has been called I copy the result in matrix
        hipMemcpy(B, d_B, n * m * sizeof(int), hipMemcpyDeviceToHost);

        double parallelExecutionTime = ((double) (clock() - t))
                        / ((double) (CLOCKS_PER_SEC));




        /////////////////////////// SECOND EXECUTION ///////////////////////////
        t = clock();

        //calling function
        parallelTransposeMemCoalescing<<<grid, threadblock>>>(d_A, d_B, m, n);
        hipDeviceSynchronize();

        //once the function has been called I copy the result in matrix
        hipMemcpy(B, d_B, n * m * sizeof(int), hipMemcpyDeviceToHost);

        double improvedParallelExecutionTime = ((double) (clock() - t))
                        / ((double) (CLOCKS_PER_SEC));





        /////////////////////////// PRINTING RESULTS ///////////////////////////

        printf("%d;%f;%f\n", m, parallelExecutionTime, improvedParallelExecutionTime);

        hipFree(d_A);
        hipFree(d_B);

        free(A);
        free(B);

        return 0;
}
